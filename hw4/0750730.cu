/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>
 #include <math.h>
 #include <time.h>
 
 #define MAXPOINTS 1000000
 #define MAXSTEPS 1000000
 #define MINPOINTS 20
 #define PI 3.14159265
 #define BLOCKSIZE 128
 
 void check_param(void);
 void init_line(void);
 void update (void);
 void printfinal (void);
 
 int nsteps,                 	/* number of time steps */
     tpoints, 	     		/* total points along string */
     rcode;                  	/* generic return code */
 float  values[MAXPOINTS+2], 	/* values at time t */
        oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
        newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
 
 
 /**********************************************************************
  *	Checks input values from parameters
  *********************************************************************/
 void check_param(void)
 {
    char tchar[20];
 
    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
       printf("Enter number of points along vibrating string [%d-%d]: "
            ,MINPOINTS, MAXPOINTS);
       scanf("%s", tchar);
       tpoints = atoi(tchar);
       if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
          printf("Invalid. Please enter value between %d and %d\n", 
                  MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
       printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
       scanf("%s", tchar);
       nsteps = atoi(tchar);
       if ((nsteps < 1) || (nsteps > MAXSTEPS))
          printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }
 
    printf("Using points = %d, steps = %d\n", tpoints, nsteps);
 
 }
 
 /**********************************************************************
  *     Initialize points on line
  *********************************************************************/
//  void init_line(void)
//  {
//     int j;
 
//     /* Calculate initial values based on sine curve */
//     for (j = 1; j <= tpoints; j++) {
//        values[j] = sin (2.0 * PI * (float)(j-1)/(tpoints-1));
//        oldval[j] = values[j];
//     }
 
//  //    /* Initialize old values array */
//  //    for (i = 1; i <= tpoints; i++) 
//  //       oldval[i] = values[i];
//  }
 
 /**********************************************************************
  *      Calculate new values using wave equation
  *********************************************************************/
 // void do_math(int i)
 // {
 //    float dtime, c, dx, tau, sqtau;
 
 //    dtime = 0.3;
 //    c = 1.0;
 //    dx = 1.0;
 //    tau = (c * dtime / dx);
 //    sqtau = tau * tau;
 //    newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
 // }
 
 /**********************************************************************
  *     Update all values along line a specified number of times
  *********************************************************************/
  __global__ void update(float *values_cuda, int tpoints,  int nsteps)
 {
     int i;
     int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
     if (j <= tpoints) {
         float values_tmp, newval_tmp, oldval_tmp;
         values_tmp = sin(2.0f * PI * (float)(j-1)/(tpoints-1));
         oldval_tmp = values_tmp;
         /* Update values for each time step */
         for (i = 1; i<= nsteps; i++) {
             /* global endpoints */
             if ((j == 1) || (j  == tpoints))
                 newval_tmp = 0.0f;
             else
                 newval_tmp = (2.0f * values_tmp) - oldval_tmp + (0.09f * (-2.0f) * values_tmp);
 
             /* Update old values with new values */
             oldval_tmp = values_tmp;
             values_tmp = newval_tmp;
         }
         values_cuda[j] = values_tmp;
     }
 }
 
 /**********************************************************************
  *     Print final results
  *********************************************************************/
 void printfinal()
 {
    int i;
 
    for (i = 1; i <= tpoints; i++) {
       printf("%6.4f ", values[i]);
       if (i%10 == 0)
          printf("\n");
    }
 }
 
 /**********************************************************************
  *	Main program
  *********************************************************************/
 int main(int argc, char *argv[])
 {
     sscanf(argv[1],"%d",&tpoints);
     sscanf(argv[2],"%d",&nsteps);
     check_param();
     printf("Initializing points on the line...\n");
     printf("Updating all points for all time steps...\n");
     float *values_cuda;
     int size = (tpoints)*sizeof(float);
     hipMalloc((void**)&values_cuda, size);
     update<<<(tpoints)/BLOCKSIZE+1, BLOCKSIZE>>>(values_cuda, tpoints, nsteps);
     hipMemcpy(values, values_cuda, size, hipMemcpyDeviceToHost);
     printf("Printing final results...\n");
     printfinal();
     printf("\nDone.\n\n");
     
     hipFree(values_cuda);
     return 0;
 }
 